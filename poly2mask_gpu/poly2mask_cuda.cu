
#include <hip/hip_runtime.h>
__global__ void poly2mask_cuda(
	int   * mask,
	int     nMaskPoints,
	int     nPolygonEdges,
	float * xs,
	float * ys,
	int     height)
{
	int idx = blockDim.x * (gridDim.x * blockIdx.y + blockIdx.x) + threadIdx.x;
	if (idx >= nMaskPoints || nPolygonEdges < 3) // At least 3 polyon points.
	{
		return;
	}

	int x = idx / height;
	int y = idx % height;
	
	float x0, y0, x1, y1;
	int wn = 0;
	
    for (int i = 0; i < nPolygonEdges; i++)
    {
		x0 = xs[i];
		y0 = ys[i];
		
		x1 = xs[i+1];
		y1 = ys[i+1];
		
		if (y0 <= y && y < y1)
		{
			if (((x1 - x0) * (y - y0) - (x - x0) * (y1 - y0)) > 0)
			{
				++wn;
			}
		}
		else if (y1 <= y && y < y0)
		{
			if (((x1 - x0) * (y - y0) - (x - x0) * (y1 - y0)) < 0)
			{
				--wn;
			}
		}
		
    }
	
	if (wn != 0)
	{
		mask[idx] = 1;
	}
}
